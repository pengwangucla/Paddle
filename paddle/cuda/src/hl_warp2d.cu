#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <cmath>
#include "hl_base.h"
#include "paddle/utils/Logging.h"


#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)


__global__ void Warp2DForward(real* input,
                              real* flow, 
                              real* output,
                              const int channel,
                              const int height,
                              const int width) {
  int nthreads = height * width;
  CUDA_KERNEL_LOOP(index, nthreads) {

      const int w = index % width;
      const int h = (index / width) % height;
      // which channel 
      const int pix_num = height * width;
      // from the 
      float xx = float(w) + flow[index] * float(width);
      float yy = float(h) + flow[index + pix_num] * float(height);
      
      float x1 = float(floorf(xx));
      float x2 = x1 + 1.0;

      float y1 = float(floorf(yy));
      float y2 = y1 + 1.0;


      if (x1 < 0. || x1 > width-1. ||
          y1 < 0. || y1 > height-1.) {
          for (int cc = 0; cc<channel; cc++){
               int off =  cc * height * width;
               output[w + h * width + off] = 0;// index of a perticular r/g/b pixel in image:  h * width + w + (n*channels +  cc)*height*width
          }
      }
      else if (x2 > width-1. || y2 > height-1. ) {
        for (int cc = 0; cc < channel; cc ++){    
          int off = cc * height * width;
          // padding with boarder value
          float val = input[int(x1) + int(y1) * width + off];
          output[w + h * width + off] = val;
        }
      }
      else {
        for (int cc = 0; cc < channel; cc ++){    
          int off = cc * height * width;
          //bilinear interplate for the new value 
          float val_up, val_down, val;
          val_up = (x2-xx) * input[int(x1) + int(y1) * width + off] + 
                   (xx-x1) * input[int(x2) + int(y1) * width + off];
          val_down = (x2-xx) * input[int(x1) + int(y2) * width + off] + 
                     (xx-x1) * input[int(x2) + int(y2) * width + off];// I_in_x_y2 /= (x2-x1);  
          val = (y2-yy) * val_up + (yy-y1) * val_down; //I_in_xy /= (y2-y1);
          output[w + h * width + off] =  val;
        }           
      }
  }
}


void hl_warp2d_forward(real *input,
                       real *flow, 
                       real *output,
                       const int channel,
                       const int height,
                       const int width) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(flow);
  const int threads = 512;
  const int blocks = DIVUP(height * width, threads);

  Warp2DForward<<<blocks, threads, 0, STREAM_DEFAULT>>>(
        input, flow, output, channel, height, width);
  CHECK_SYNC("hl_warp2d_forward failed");
}


__device__ inline void Angle2Matrix(const real* ang,
                                    real* R) {
    real angle = sqrtf(ang[0] * ang[0] + 
                   ang[1] * ang[1] + 
                   ang[2] * ang[2]);

    if( angle > real(1e-6) )
    {
      real c = cosf(angle);
      real s = sinf(angle);
      real u[3] = {ang[0]/angle, ang[1]/angle, ang[2]/angle};

      R[0] = c+u[0]*u[0]*(1-c);      
      R[3] = u[1]*u[0]*(1-c)+u[2]*s; 
      R[6] = u[2]*u[0]*(1-c)-u[1]*s; 

      R[1] = u[0]*u[1]*(1-c)-u[2]*s; 
      R[4] = c+u[1]*u[1]*(1-c);      
      R[7] = u[2]*u[1]*(1-c)+u[0]*s; 

      R[2] = u[0]*u[2]*(1-c)+u[1]*s;
      R[5] = u[1]*u[2]*(1-c)-u[0]*s;
      R[8] = c+u[2]*u[2]*(1-c);
    }
    else
    {
      R[0] = 1; R[3] = 0; R[6] = 0;
      R[1] = 0; R[4] = 1; R[7] = 0;
      R[2] = 0; R[5] = 0; R[8] = 1;
    }
}

// trans: [fx, fy, ux, uy, a1, a2, a3, t1, t2, t3]
__global__ void Depth2FlowForward(real* depth,
                              real* trans, 
                              real* flow,
                              const int batch_size,
                              const int height,
                              const int width) {

  int nthreads = height * width * batch_size;
  CUDA_KERNEL_LOOP(index, nthreads) {

    // transfer depth to 3d
    real x = real(index % width);
    real y = real((index / width) % height);
    int batch_id = index / (height * width);
    int image_size = height * width;

    real* cur_trans = trans + 10 * batch_id;

    real* f = cur_trans;
    real* u = cur_trans + 2;
    real* r = cur_trans + 4;
    real* t = cur_trans + 7;

    
    real x_3d[3] = {0.0f, 0.0f, 0.0f};
    x_3d[0] = (x - u[0]) / f[0] * depth[index];
    x_3d[1] = (y - u[1]) / f[1] * depth[index];
    x_3d[2] = depth[index];

    real R[9];
    Angle2Matrix(r, R);

    // project 3d to the second image
    real x_tmp_3d[3] = {0.f, 0.f, 0.f};
    for(int i = 0; i < 3; i ++)
      for(int j = 0; j < 3; j ++)
        x_tmp_3d[i] += R[i*3 + j] * x_3d[j];

    for(int i = 0; i < 3; i ++) x_3d[i] = x_tmp_3d[i];
    for(int i = 0; i < 3; i ++) x_3d[i] += t[i];

    // calculate the flow
    real x2 = x_3d[2] == 0. ? 0. : ((x_3d[0] / x_3d[2] * f[0]) + u[0]);
    real y2 = x_3d[2] == 0. ? 0. : ((x_3d[1] / x_3d[2] * f[1]) + u[1]);

    flow[batch_id * 2 * image_size + int(y) * width + int(x)] = 
                                      x_3d[2] == 0 ? 0. : x2 - x;
    flow[(batch_id * 2 + 1) * image_size + int(y) * width + int(x)] = 
                                      x_3d[2] == 0 ? 0 : y2 - y;
  }
}


__device__ void MatMultiply(real* A, size_t row_A, size_t col_A,
                            real* B, size_t row_B, size_t col_B,
                            real* C) {
  for(int i = 0; i < row_A; i ++)
    for(int m = 0; m < col_B; m ++) {
      C[i * col_B + m] = 0.0f;
      for(int j = 0; j < col_A; j ++) {
          C[i * col_B + m] += A[i * col_A + j] * B[j * col_B + m];
      }
    }
}


// trans: [fx, fy, ux, uy, a1, a2, a3, t1, t2, t3]
__global__ void Flow2DepthForward(real* flow,
                              real* trans, 
                              real* depth,
                              const int batch_size,
                              const int height,
                              const int width) {
    // project 3d to the second image
  int nthreads = height * width * batch_size;
  CUDA_KERNEL_LOOP(index, nthreads) {
    real x = real(index % width),
         y = real((index / width) % height);

    int batch_id = index / (height * width);
    int image_size = height * width;

    real x2 = x + 
      flow[batch_id * 2 * image_size + int(y) * width + int(x)];
    real y2 = y + 
      flow[(batch_id * 2 + 1) * image_size + int(y) * width + int(x)];

    real* cur_trans = trans + 10 * batch_id;
    real* f = cur_trans;
    real* u = cur_trans + 2;
    real* r = cur_trans + 4;
    real* t = cur_trans + 7;

    real x_3d[3] = {0.0f, 0.0f, 1.0f};
    x_3d[0] = (x - u[0]) / f[0];
    x_3d[1] = (y - u[1]) / f[1];

    real x2_3d[2] = {0.0f, 0.0f};
    x2_3d[0] = (x2 - u[0]) / f[0];
    x2_3d[1] = (y2 - u[1]) / f[1];

    real R[9];
    Angle2Matrix(r, R);

    real res[3];
    MatMultiply(R, 3, 3, x_3d, 3, 1, res);

    real depth_cur = 0.0f;
    real counter = 0.0f;
    real div = res[2] * x2_3d[1] - res[1];
    if( div != 0) {
      depth_cur += (t[1] - x2_3d[1] * t[2]) / div;
      counter += 1.0f;
    }
    div = res[2] * x2_3d[0] - res[0];
    if(div != 0) {
      depth_cur += (t[0] - x2_3d[0] * t[2]) / div;
      counter += 1.0f;
    }
    depth[index] = depth_cur / max(counter, real(1e-6));
    
  }
}


// trans: [fx, fy, ux, uy, a1, a2, a3, t1, t2, t3]
void hl_trans_depth_flow_forward(real *input,
                       real *trans, 
                       real *output,
                       const int batch_size,
                       const int height,
                       const int width,
                       const bool depth_to_flow) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(trans);

  const int threads = 1024;
  const int blocks = DIVUP(height * width * batch_size, threads);
  if(depth_to_flow) {
    Depth2FlowForward<<<blocks, threads, 0, STREAM_DEFAULT>>>(
          input, trans, output, batch_size, height, width);
  }
  else {
    Flow2DepthForward<<<blocks, threads, 0, STREAM_DEFAULT>>>(
          input, trans, output, batch_size, height, width);
  }

  CHECK_SYNC("hl_trans_depth_flow_forward failed");
}