#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <cmath>
#include "hl_base.h"
#include "paddle/utils/Logging.h"


#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)


__global__ void Warp2DForward(real* input,
                              real* flow, 
                              real* output,
                              const int channel,
                              const int height,
                              const int width) {
  int nthreads = height * width;
  CUDA_KERNEL_LOOP(index, nthreads) {

      const int w = index % width;
      const int h = (index / width) % height;
      // which channel 
      const int pix_num = height * width;
      // from the 
      float xx = float(w) + flow[index];
      float yy = float(h) + flow[index + pix_num];
      
      float x1 = float(floorf(xx));
      float x2 = x1 + 1.0;

      float y1 = float(floorf(yy));
      float y2 = y1 + 1.0;


      if (x1 < 0. || x1 > width-1. ||
          y1 < 0. || y1 > height-1.) {
          for (int cc = 0; cc<channel; cc++){
               int off =  cc * height * width;
               output[w + h * width + off] = 0;// index of a perticular r/g/b pixel in image:  h * width + w + (n*channels +  cc)*height*width
          }
      }
      else if (x2 > width-1. || y2 > height-1. ) {
        for (int cc = 0; cc < channel; cc ++){    
          int off = cc * height * width;
          // padding with boarder value
          float val = input[int(x1) + int(y1) * width + off];
          output[w + h * width + off] = val;
        }
      }
      else {
        for (int cc = 0; cc < channel; cc ++){    
          int off = cc * height * width;
          //bilinear interplate for the new value 
          float val_up, val_down, val;
          val_up = (x2-xx) * input[int(x1) + int(y1) * width + off] + 
                   (xx-x1) * input[int(x2) + int(y1) * width + off];
          val_down = (x2-xx) * input[int(x1) + int(y2) * width + off] + 
                     (xx-x1) * input[int(x2) + int(y2) * width + off];// I_in_x_y2 /= (x2-x1);  
          val = (y2-yy) * val_up + (yy-y1) * val_down; //I_in_xy /= (y2-y1);
          output[w + h * width + off] =  val;
        }           
      }
  }
}


void hl_warp2d_forward(real *input,
                       real *flow, 
                       real *output,
                       const int channel,
                       const int height,
                       const int width) {
  CHECK_NOTNULL(input);
  CHECK_NOTNULL(flow);
  const int threads = 512;
  const int blocks = DIVUP(height * width, threads);

  Warp2DForward<<<blocks, threads, 0, STREAM_DEFAULT>>>(
        input, flow, output, channel, height, width);
  CHECK_SYNC("hl_warp2d_forward failed");
}

